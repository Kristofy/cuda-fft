#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include "image.h"


__global__ void grayscale_kernel(int* d_input_image, int* d_output_image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int idx = y * width + x;
        int r = d_input_image[3 * idx + 0];
        int g = d_input_image[3 * idx + 1];
        int b = d_input_image[3 * idx + 2];
        int color = 0.299f * r + 0.587f * g + 0.114f * b;
        d_output_image[3 * idx + 0] = color;
        d_output_image[3 * idx + 1] = color;
        d_output_image[3 * idx + 2] = color;
    }
}

__global__ void split_channels_kernel(Pixel* d_input_pixels, int width, int height, double* output_data_r, double* output_data_g, double* output_data_b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < width && idy < height) {
        int offset = idy * width + idx;
        output_data_r[offset] = d_input_pixels[offset].r;
        output_data_g[offset] = d_input_pixels[offset].g;
        output_data_b[offset] = d_input_pixels[offset].b;
    }
}

__global__ void merge_channels_kernel(Pixel* d_input_pixels, int width, int height, hipfftDoubleComplex* d_input_data_r, hipfftDoubleComplex* d_input_data_g, hipfftDoubleComplex* d_input_data_b) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int index = y * width + x;
        int i = 0;
        if(y < height && x * height <= width * y) {
            i = (y - 1) * width * y / (2 * height) + x;
        } else {
            int dy = height - y - 1;
            int dx = width - x - 1;
            i = (dy - 1) * width * dy / (2 * height) + dx;
        }
        d_input_pixels[index].r = (int) (std::sqrt(d_input_data_r[i].x * d_input_data_r[i].x + d_input_data_r[i].y * d_input_data_r[i].y) * 255 / (width * height));
        d_input_pixels[index].g = (int) (std::sqrt(d_input_data_g[i].x * d_input_data_g[i].x + d_input_data_g[i].y * d_input_data_g[i].y) * 255 / (width * height));
        d_input_pixels[index].b = (int) (std::sqrt(d_input_data_b[i].x * d_input_data_b[i].x + d_input_data_b[i].y * d_input_data_b[i].y) * 255 / (width * height));
    }
}


template<typename KernelT>
__host__ void run_kernel_on_image(Image& img, KernelT k){
    
    int* d_input_image;
    int* d_output_image;
    size_t image_size_bytes = img.width * img.height * 3 * sizeof(int);

    hipMalloc(&d_input_image, image_size_bytes);
    hipMalloc(&d_output_image, image_size_bytes);

    hipMemcpy(d_input_image, img.pixels, image_size_bytes, hipMemcpyHostToDevice);

    int block_size = 32;
    dim3 dim_grid((img.width + block_size - 1) / block_size, (img.height + block_size - 1) / block_size);
    dim3 dim_block(block_size, block_size);

    k<<<dim_grid, dim_block>>>(d_input_image, d_output_image, img.width, img.height);

    
    hipMemcpy(img.pixels, d_output_image, image_size_bytes, hipMemcpyDeviceToHost);

    hipFree(d_input_image);
    hipFree(d_output_image);
}

__host__ void run_fft_on_imge(Image& img){
    const int width = img.width;
    const int height = img.height;

    double* d_input_data_r;
    double* d_input_data_g;
    double* d_input_data_b;
    hipMalloc((void**)&d_input_data_r, sizeof(double) * width * height);
    hipMalloc((void**)&d_input_data_g, sizeof(double) * width * height);
    hipMalloc((void**)&d_input_data_b, sizeof(double) * width * height);

    hipfftDoubleComplex* d_output_data_r;
    hipfftDoubleComplex* d_output_data_g;
    hipfftDoubleComplex* d_output_data_b;
    hipMalloc((void**)&d_output_data_r, sizeof(hipfftDoubleComplex) * width * (height / 2 + 1));
    hipMalloc((void**)&d_output_data_g, sizeof(hipfftDoubleComplex) * width * (height / 2 + 1));
    hipMalloc((void**)&d_output_data_b, sizeof(hipfftDoubleComplex) * width * (height / 2 + 1));

    Pixel *d_input_pixels;
    hipMalloc((void**)&d_input_pixels, sizeof(Pixel) * width * height);

    hipMemcpy(d_input_pixels, img.pixels, sizeof(Pixel) * width * height, hipMemcpyHostToDevice);
    
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
   
    split_channels_kernel<<<gridSize, blockSize>>>(d_input_pixels, width, height, d_input_data_r, d_input_data_g, d_input_data_b);

    hipfftHandle plan_r, plan_g, plan_b;
    hipfftPlan2d(&plan_r, width, height, HIPFFT_D2Z);
    hipfftPlan2d(&plan_g, width, height, HIPFFT_D2Z);
    hipfftPlan2d(&plan_b, width, height, HIPFFT_D2Z);

    hipfftExecD2Z(plan_r, d_input_data_r, (hipfftDoubleComplex*)d_output_data_r);
    hipfftExecD2Z(plan_g, d_input_data_g, (hipfftDoubleComplex*)d_output_data_g);
    hipfftExecD2Z(plan_b, d_input_data_b, (hipfftDoubleComplex*)d_output_data_b);

    merge_channels_kernel<<<gridSize, blockSize>>>(d_input_pixels, width, height, d_output_data_r, d_output_data_g, d_output_data_b);

    hipMemcpy(img.pixels, d_input_pixels, sizeof(Pixel) * width * height, hipMemcpyDeviceToHost);

    hipFree(d_input_data_r);
    hipFree(d_input_data_g);
    hipFree(d_input_data_b);
    hipFree(d_output_data_r);
    hipFree(d_output_data_g);
    hipFree(d_output_data_b);
    hipFree(d_input_pixels);
    hipfftDestroy(plan_r);
    hipfftDestroy(plan_g);
    hipfftDestroy(plan_b);
}


int main(int argc, char **argv) {
    // Load input image data
    Image img(argv[1]);
    // int width = img.width;
    // int height = img.height;
    // run_kernel_on_image(img, grayscale_kernel);
    run_fft_on_imge(img);
    // run_kernel_on_image(img, grayscale_kernel);

    // img.WriteImage(std::string(argv[2])+"_gray");


    // double* h_input_data = (double*)malloc(sizeof(double) * width * height); // Load input image data
    // for(int i = 0; i < width*height; i++){
    //     h_input_data[i] = img.pixels[i].r ;
    // }

    // // Execute 2D FFT
    // fft2d(h_input_data, width, height, h_input_data);

    // for(int i = 0; i < width * height; i++){
    //     img.pixels[i] = {(int)h_input_data[i],(int)h_input_data[i],(int)h_input_data[i]};
    // }

    img.WriteImage(argv[2]);
    // Free memory
    // free(h_input_data);

    return 0;
}
